/**
 * cnf_formula_gpu.cu: definition of the CNF_Formula device API.
 * 
 * Copyright (c) Michele Collevati
 */


#include <stdlib.h>
#include <stdio.h>


#include "cnf_formula_gpu.cuh"
#include "utils.cuh"


/**
 * API definition
 */


CNF_Formula *cnf_gpu_transfer_formula_host_to_dev(CNF_Formula *phi) {
    CNF_Formula *d_phi;
    gpuErrchk( hipMalloc((void**)&d_phi, sizeof *d_phi) );
    gpuErrchk( hipMemcpy(d_phi, phi, sizeof *d_phi, hipMemcpyHostToDevice) );
    
    Lidx *d_clauses;
    int clauses_len = phi->clauses_len;
    gpuErrchk( hipMalloc((void**)&d_clauses,
                          sizeof *d_clauses * clauses_len) );
    gpuErrchk( hipMemcpy(&(d_phi->clauses), &d_clauses,
                          sizeof d_clauses,
                          hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_clauses, phi->clauses,
                          sizeof *d_clauses * clauses_len,
                          hipMemcpyHostToDevice) );
    
    int *d_clause_indices;
    int clause_indices_len = phi->clause_indices_len;
    gpuErrchk( hipMalloc((void**)&d_clause_indices,
                          sizeof *d_clause_indices *
                          clause_indices_len) );
    gpuErrchk( hipMemcpy(&(d_phi->clause_indices), &d_clause_indices,
                          sizeof d_clause_indices,
                          hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_clause_indices, phi->clause_indices,
                          sizeof *d_clause_indices * clause_indices_len,
                          hipMemcpyHostToDevice) );

    return d_phi;
}


CNF_Formula *cnf_gpu_transfer_formula_dev_to_host(CNF_Formula *d_phi) {
    CNF_Formula *phi = (CNF_Formula *)malloc(sizeof *phi);
    gpuErrchk( hipMemcpy(phi, d_phi, sizeof *phi, hipMemcpyDeviceToHost) );

    int clauses_len = phi->clauses_len;
    phi->clauses = (Lidx *)malloc(sizeof *(phi->clauses) * clauses_len);
    Lidx *d_clauses;
    gpuErrchk( hipMemcpy(&d_clauses, &(d_phi->clauses),
                          sizeof d_clauses,
                          hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(phi->clauses, d_clauses,
                          sizeof *d_clauses * clauses_len,
                          hipMemcpyDeviceToHost) );

    int clause_indices_len = phi->clause_indices_len;
    phi->clause_indices = (int *)malloc(sizeof *(phi->clause_indices) *
                                        clause_indices_len);
    int *d_clause_indices;
    gpuErrchk( hipMemcpy(&d_clause_indices, &(d_phi->clause_indices),
                          sizeof d_clause_indices,
                          hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(phi->clause_indices, d_clause_indices,
                          sizeof *d_clause_indices * clause_indices_len,
                          hipMemcpyDeviceToHost) );

    return phi;
}


void cnf_gpu_destroy_formula(CNF_Formula *d_phi) {
    Lidx *d_clauses;
    gpuErrchk( hipMemcpy(&d_clauses, &(d_phi->clauses),
                          sizeof d_clauses,
                          hipMemcpyDeviceToHost) );
    gpuErrchk( hipFree(d_clauses) );

    int *d_clause_indices;
    gpuErrchk( hipMemcpy(&d_clause_indices, &(d_phi->clause_indices),
                          sizeof d_clause_indices,
                          hipMemcpyDeviceToHost) );
    gpuErrchk( hipFree(d_clause_indices) );

    gpuErrchk( hipFree(d_phi) );
}
