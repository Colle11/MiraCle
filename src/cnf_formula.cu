#include "hip/hip_runtime.h"
/**
 * cnf_formula.cu: definition of the CNF_Formula API.
 * 
 * Copyright (c) Michele Collevati
 */


#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>


#include "cnf_formula.cuh"
#include "utils.cuh"


// #define CHECK_VARS_IN_CLAUSE


#define AVG_NUM_LITS_IN_CLAUSE (3)    // Average number of literals in a clause.


#ifdef CHECK_VARS_IN_CLAUSE
bool repeated_var_in_clause = false;
#endif


/**
 * Auxiliary function prototypes
 */


/**
 * @brief Skips a line in a file.
 * 
 * @param [in]fp A file pointer.
 * @retval None.
 */
static void skip_line(FILE *fp);


/**
 * @brief Parses the problem line in a file in DIMACS CNF format.
 * 
 * @param [in]fp A DIMACS CNF file pointer.
 * @param [out]num_vars The number of variables declared in the problem line.
 * @param [out]num_clauses The number of clauses declared in the problem line.
 * @retval None.
 */
static void parse_problem_line(FILE *fp,
                               int *num_vars,
                               int *num_clauses);


/**
 * @brief Creates an empty formula.
 * 
 * @param [in]num_clauses A number of clauses.
 * @param [in]est_num_lits An estimate of the number of literals.
 * @retval An empty formula.
 */
static CNF_Formula *create_formula(int num_clauses,
                                   int est_num_lits);


/**
 * @brief Parses a clause line in a file in DIMACS CNF format.
 * 
 * @param [in]fp A DIMACS CNF file pointer.
 * @param [in/out]est_num_lits An estimate of the number of literals.
 * @param [in/out]phi A formula being initialized.
 * @retval None.
 */
static void parse_clause_line(FILE *fp,
                              int *est_num_lits,
                              CNF_Formula *phi);


/**
 * API definition
 */


CNF_Formula *cnf_parse_DIMACS(char *filename) {
    FILE *fp = fopen(filename, "r");

    if (fp == NULL) {
        fprintf(stderr, "Can't open %s.\n", filename);
        exit(EXIT_FAILURE);
    }

    int c;      // Currently read character.
    bool p_ln_found = false;    /**
                                 * Flag to check if the problem line has been
                                 * found.
                                 */
    int p_num_vars;             /**
                                 * Number of variables declared in the problem
                                 * line.
                                 */
    int p_num_clauses;          /**
                                 * Number of clauses declared in the problem
                                 * line.
                                 */
    int est_num_lits;           // Estimate of the number of literals.
    CNF_Formula *phi;

    while (!feof(fp)) {
        c = fgetc(fp);

        if (c == 'c') {     // Skip a comment line.
            skip_line(fp);
        } else if (c == 'p') {      // Parse the problem line.
            p_ln_found = true;

            parse_problem_line(fp, &p_num_vars, &p_num_clauses);

            est_num_lits = p_num_clauses * AVG_NUM_LITS_IN_CLAUSE;

            phi = create_formula(p_num_clauses, est_num_lits);
        } else if (('1' <= c && c <= '9') || c == '-') {
            if (!p_ln_found) {
                fprintf(stderr, "The problem line is missing in the DIMACS "
                        "CNF file \"%s\".\n", filename);
                exit(EXIT_FAILURE);
            }

            ungetc(c, fp);

            // Parse a clause line.
            parse_clause_line(fp, &est_num_lits, phi);
        }
    }

    phi->clause_indices[phi->num_clauses] = phi->num_lits;

    // Adapt the memory of phi->clauses.
    phi->clauses = (Lidx *)realloc(phi->clauses,
                                   sizeof *(phi->clauses) * phi->num_lits);
    phi->clauses_len = phi->num_lits;

    fclose(fp);

    // Check the correctness of the problem line.
    if (p_num_vars != phi->num_vars || p_num_clauses != phi->num_clauses) {
        fprintf(stderr, "The problem line of the DIMACS CNF file \"%s\" is "
                "incorrect.\n", filename);
        exit(EXIT_FAILURE);
    }

#ifdef CHECK_VARS_IN_CLAUSE
    if (repeated_var_in_clause) {
        fprintf(stderr, "Found repeated variables in the clauses of the "
                "DIMACS CNF file \"%s\".\n", filename);
        exit(EXIT_FAILURE);
    } else {
        // exit(EXIT_SUCCESS);
    }
#endif

    return phi;
}


void cnf_destroy_formula(CNF_Formula *phi) {
    free(phi->clauses);
    free(phi->clause_indices);
    free(phi);
}


void cnf_print_formula(CNF_Formula *phi) {
    printf("*** CNF formula ***\n\n");

    printf("Number of variables: %d\n", phi->num_vars);
    printf("Number of clauses: %d\n", phi->num_clauses);
    printf("Number of literals: %d\n", phi->num_lits);

    printf("Clause indices: ");
    for (int i = 0; i < phi->clause_indices_len; i++) {
        printf("[%d]%d ", i, phi->clause_indices[i]);
    }
    printf("\n");

    printf("Clauses: ");
    for (int l = 0; l < phi->clauses_len; l++) {
        printf("[%d]%d ", l, phi->clauses[l]);
    }
    printf("\n");

    printf("\n*** End CNF formula ***\n\n");
}


/**
 * Auxiliary function definitions
 */


static void skip_line(FILE *fp) {
    int c = fgetc(fp);

    while (c != '\n') {
        c = fgetc(fp);
    }
}


static void parse_problem_line(FILE *fp,
                               int *num_vars,
                               int *num_clauses) {
    fscanf(fp, "%*s %d %d", num_vars, num_clauses);

    int c = fgetc(fp);

    while (c != '\n') {
        c = fgetc(fp);
    }
}


static CNF_Formula *create_formula(int num_clauses,
                                   int est_num_lits) {
    CNF_Formula *phi = (CNF_Formula *)malloc(sizeof *phi);

    phi->clause_indices_len = num_clauses + 1;
    phi->clause_indices = (int *)malloc(sizeof *(phi->clause_indices) *
                                        phi->clause_indices_len);

    phi->clauses_len = est_num_lits;
    phi->clauses = (Lidx *)malloc(sizeof *(phi->clauses) * phi->clauses_len);

    phi->num_vars = 0;
    phi->num_clauses = 0;
    phi->num_lits = 0;

    return phi;
}


static void parse_clause_line(FILE *fp,
                              int *est_num_lits,
                              CNF_Formula *phi) {
    phi->clause_indices[phi->num_clauses] = phi->num_lits;
    phi->num_clauses++;

    bool polarity = true;
    Var var = 0;
    int c = fgetc(fp);

#ifdef CHECK_VARS_IN_CLAUSE
    int num_lits_in_clause = 0;
#endif

    while (c != '0') {
        if (c == '-') {
            polarity = false;
        } else {    // '1' <= c && c <= '9'
            var = c - '0';
        }

        c = fgetc(fp);

        while ('0' <= c && c <= '9') {      // Parse a literal.
            var *= 10;
            var += c - '0';
            c = fgetc(fp);
        }

#ifdef CHECK_VARS_IN_CLAUSE
        for (int l = 1; l <= num_lits_in_clause; l++) {
            if (var - 1 == lidx_to_var(phi->clauses[phi->num_lits - l])) {
                fprintf(stderr,
                        "Clause: %d\tVar: %d\n",
                        phi->num_clauses,
                        var);
                repeated_var_in_clause = true;
            }
        }

        num_lits_in_clause++;
#endif

        phi->num_vars = max(phi->num_vars, var);

        // Increase the memory of phi->clauses.
        if (phi->num_lits == *est_num_lits) {
            *est_num_lits *= 2;
            
            phi->clauses = (Lidx *)realloc(phi->clauses,
                                           sizeof *(phi->clauses) *
                                           (*est_num_lits));
            phi->clauses_len = *est_num_lits;
        }

        phi->clauses[phi->num_lits] = varpol_to_lidx(var - 1, polarity);
        phi->num_lits++;

        var = 0;
        polarity = true;

        while (!('1' <= c && c <= '9') && c != '-' && c != '0') {
            c = fgetc(fp);
        }
    }
}
